#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "opencv2/calib3d.hpp"
#include "opencv2/imgproc.hpp"
#include "opencv2/imgcodecs.hpp"
#include "opencv2/highgui.hpp"
#include "opencv2/core/utility.hpp"
#include <iostream>
#include <bitset>
#include "./helpers/helper.cuh"
#include "./calc_cost/calc_cost_and_disparity.cuh"
#include "./calc_path/calc_path.cuh"

using namespace cv;

using namespace cv::cuda;
using namespace std;

#define D_LVL 64

#define checkCudaErrors(call)                                 \
  do {                                                        \
    hipError_t err = call;                                   \
    if (err != hipSuccess) {                                 \
      printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, \
             hipGetErrorString(err));                        \
      exit(EXIT_FAILURE);                                     \
    }                                                         \
  } while (0)



__host__ void allProcessOnCUDA(
    unsigned char* census_l_R, unsigned char* census_l_G, unsigned char* census_l_B,
    unsigned char* census_r_R, unsigned char* census_r_G, unsigned char* census_r_B,
    uchar* disparityMap,
    size_t rows, size_t cols
) {
    int numBytes = rows * cols * D_LVL * sizeof(int);
    int smallBytes = rows * cols * D_LVL * sizeof(unsigned char);

    // allocate device memory
    unsigned char * adev = NULL, *bdev = NULL, *disparityRes = NULL;
    int * extraStore = NULL;
    int * resCuda = NULL, *middleRes = NULL;

    checkCudaErrors(hipMalloc ( (void**)&adev, smallBytes ));
    checkCudaErrors(hipMalloc ( (void**)&bdev, smallBytes ));
    checkCudaErrors(hipMalloc ( (void**)&middleRes, numBytes ));
    checkCudaErrors(hipMalloc ( (void**)&extraStore, numBytes ));
    checkCudaErrors(hipMalloc ( (void**)&resCuda, numBytes  ));
    checkCudaErrors(hipMalloc ( (void**)&disparityRes, smallBytes));

    // set kernel launch configuration
  
    dim3 threads ( D_LVL );
    dim3 blocks  ( rows, cols );
    // create cuda event handles
    hipEvent_t start, stop;
    float gpuTime;
    float allRes = 0;
    int countCheck = 1;

    // TimeCheck
    for (int i = 0; i< countCheck ; i++) {
        gpuTime = 0.0f;
        checkCudaErrors(hipEventCreate ( &start ));
        checkCudaErrors(hipEventCreate ( &stop ));
        
        // asynchronously issue work to the GPU (all to stream 0)
        hipEventRecord ( start,  0 );

        clearResCUDA<<<blocks, threads>>> (middleRes, rows, cols);
        clearResCUDA<<<blocks, threads>>> (resCuda, rows, cols);
        clearResCUDA<<<blocks, threads>>> (disparityRes, rows, cols);
        // COST
        checkCudaErrors(hipMemcpy( adev, census_l_R, smallBytes, hipMemcpyHostToDevice ));
        checkCudaErrors(hipMemcpy( bdev, census_r_R, smallBytes, hipMemcpyHostToDevice ));  
        calculateInitialCostCUDA <<<blocks, threads>>> ( adev, bdev, extraStore, rows, cols);
        optimisedConcatResCUDA<<<blocks, threads>>> (extraStore, middleRes, rows, cols);

        checkCudaErrors(hipMemcpy( adev, census_l_G, smallBytes, hipMemcpyHostToDevice ));
        checkCudaErrors(hipMemcpy( bdev, census_r_G, smallBytes, hipMemcpyHostToDevice ));  
        calculateInitialCostCUDA <<<blocks, threads>>> ( adev, bdev, extraStore, rows, cols);
        optimisedConcatResCUDA<<<blocks, threads>>> (extraStore, middleRes, rows, cols);
    
        checkCudaErrors(hipMemcpy( adev, census_l_B, smallBytes, hipMemcpyHostToDevice ));
        checkCudaErrors(hipMemcpy( bdev, census_r_B, smallBytes, hipMemcpyHostToDevice ));  
        calculateInitialCostCUDA <<<blocks, threads>>> ( adev, bdev, extraStore, rows, cols);
        optimisedConcatResCUDA<<<blocks, threads>>> (extraStore, middleRes, rows, cols);


        // PATH
        calculatePathLeft<<<rows, D_LVL>>> ( middleRes, extraStore, rows, cols);
        optimisedConcatResCUDA<<<blocks, threads>>> (extraStore, resCuda, rows, cols);
        calculatePathRight<<<rows, D_LVL>>> (middleRes, extraStore, rows, cols);
        optimisedConcatResCUDA<<<blocks, threads>>> (extraStore, resCuda, rows, cols);
        calculatePathTop<<<cols, D_LVL>>> (middleRes, extraStore, rows, cols);
        optimisedConcatResCUDA<<<blocks, threads>>> (extraStore, resCuda, rows, cols);
        calculatePathBackslash<<<cols + rows - 1, D_LVL>>> (middleRes, extraStore, rows, cols);
        optimisedConcatResCUDA<<<blocks, threads>>> (extraStore, resCuda, rows, cols);
        calculatePathSlash<<<cols + rows - 1, D_LVL>>> (middleRes, extraStore, rows, cols);
        optimisedConcatResCUDA<<<blocks, threads>>> (extraStore, resCuda, rows, cols);


        //Disparyty
        calculateDisparityCUDA<<<blocks, 1>>> (resCuda, disparityRes, cols);

        checkCudaErrors(hipMemcpy( disparityMap, disparityRes, smallBytes, hipMemcpyDeviceToHost ));
        
        hipEventRecord ( stop, 0 );

        hipEventSynchronize ( stop );
        hipEventElapsedTime ( &gpuTime, start, stop );

        hipEventDestroy ( start );
        hipEventDestroy ( stop  );
        allRes += gpuTime;
       
        // printf("Time spent executing by the GPU: %.3f millseconds\n", gpuTime);
    }
    
    printf("Average Time spent executing by the GPU: %.3f millseconds for COUNT=%d \n", allRes / countCheck, countCheck);

    // release resources
    checkCudaErrors(hipFree( adev  ));
    hipFree  ( bdev );
    hipFree  ( middleRes );
    hipFree  ( extraStore );
    hipFree  ( resCuda );
    hipFree  ( disparityRes );
}


cv::Mat* calculateImageDisparity(cv::Mat &leftImage, cv::Mat &rightImage) {
    double costTime;
    size_t cols = leftImage.cols, rows = leftImage.rows;
    uchar *disparityMap = (uchar *) calloc(rows * cols * D_LVL, sizeof(uchar));

    if (!disparityMap) {
        printf("mem failure, exiting A \n");
        exit(EXIT_FAILURE);
    }


    Mat splitResult[3], splitResultRight[3];
    split(leftImage, splitResult);
    Mat leftImageR = splitResult[0];
    Mat leftImageG = splitResult[1];
    Mat leftImageB = splitResult[2];
    split(rightImage, splitResultRight);
    Mat rightImageR = splitResultRight[0];
    Mat rightImageG = splitResultRight[1];
    Mat rightImageB = splitResultRight[2];

    // imshow("leftImageR", leftImageR);
    // imshow("leftImageG", leftImageG);
    // imshow("leftImageB", leftImageB);

    unsigned char *census_l_R = (unsigned char *) calloc(rows * cols * D_LVL, sizeof(unsigned char));
    unsigned char *census_l_G = (unsigned char *) calloc(rows * cols * D_LVL, sizeof(unsigned char));
    unsigned char *census_l_B = (unsigned char *) calloc(rows * cols * D_LVL, sizeof(unsigned char));
    unsigned char *census_r_R = (unsigned char *) calloc(rows * cols * D_LVL, sizeof(unsigned char));
    unsigned char *census_r_G = (unsigned char *) calloc(rows * cols * D_LVL, sizeof(unsigned char));
    unsigned char *census_r_B = (unsigned char *) calloc(rows * cols * D_LVL, sizeof(unsigned char));

    // 1. Census Transform"
    census_transform(leftImageR, census_l_R, rows, cols);
    census_transform(leftImageG, census_l_G, rows, cols);
    census_transform(leftImageB, census_l_B, rows, cols);
    census_transform(rightImageR, census_r_R, rows, cols);
    census_transform(rightImageG, census_r_G, rows, cols);
    census_transform(rightImageB, census_r_B, rows, cols);

    // 2. Calculate Pixel Cost.
    // 3. Aggregate Cost
    // 4. Create Disparity Image.
    //One CUDA operation
    costTime = (double) getTickCount();
    allProcessOnCUDA(census_l_R, census_l_G, census_l_B, census_r_R, census_r_G, census_r_B, disparityMap, rows, cols);
    costTime = ((double)getTickCount() - costTime)/getTickFrequency();

    cout<<"Cost algorithm time: "<< costTime <<"s"<<endl;  // 123ms

    free(census_l_R);
    free(census_r_G);
    free(census_l_B);
    free(census_r_R);
    free(census_l_G);
    free(census_r_B);
    
    return new Mat(rows, cols, CV_8UC1, disparityMap);
}


int main () {
    double solving_time, allTimeSolving = (double) getTickCount();
    // Mat leftImage = cv::imread("./src/images/leftImage1.png",cv::IMREAD_GRAYSCALE);
    // Mat rightImage = cv::imread("./src/images/rightImage1.png",cv::IMREAD_GRAYSCALE);
    // Mat leftImage = cv::imread("./src/images/leftImage1.png",cv::IMREAD_COLOR);
    // Mat rightImage = cv::imread("./src/images/rightImage1.png",cv::IMREAD_COLOR);
    // Mat leftImage = cv::imread("./src/images/appleLeft.jpg");
    // Mat rightImage = cv::imread("./src/images/appleRight.jpg");
    // Mat leftImage = cv::imread("./src/images/warLeft.jpg",cv::IMREAD_COLOR);
    // Mat rightImage = cv::imread("./src/images/warRight.jpg",cv::IMREAD_COLOR);
    Mat leftImage = cv::imread("./src/images/warLeft.jpg");
    Mat rightImage = cv::imread("./src/images/warRight.jpg");
    // Mat leftImage = cv::imread("./src/images/warLeft.jpg",cv::IMREAD_GRAYSCALE);
    // Mat rightImage = cv::imread("./src/images/warRight.jpg",cv::IMREAD_GRAYSCALE);

    // imshow("leftImage", leftImage);
    // imshow("rightImage", rightImage);

    cv::Mat *disparityMap;

    cout.precision(3);
    cout << " Start timing"<< endl;
    solving_time = (double) getTickCount();

    // resize(leftImage, left_for_matcher, Size(),0.1,0.1, INTER_LINEAR_EXACT);
    // cvtColor(left_for_matcher,  left_for_matcher,  COLOR_BGR2GRAY);
    // left_for_matcher.convertTo(left_for_matcher, CV_16UC1);
    // leftImage.convertTo(leftImage, CV_16UC3);
    disparityMap = calculateImageDisparity(leftImage, rightImage);




    // END TEST GRAYSCALE
    solving_time = ((double)getTickCount() - solving_time)/getTickFrequency();
 

    // Visualize Disparity Image.
    disparityMap->convertTo(*disparityMap, CV_8U, 256.0/D_LVL);
    applyColorMap(*disparityMap, *disparityMap, COLORMAP_JET);
    imshow("disparityMap", *disparityMap);

    allTimeSolving = ((double)getTickCount() - allTimeSolving)/getTickFrequency();
    cout<<"Process time: "<<solving_time<<"s"<<endl;     // 179ms
    cout<<"All run time: "<<allTimeSolving<<"s"<<endl;   // 184ms
    std::cout << "OK"<< std::endl;

    free(disparityMap);

    while(1)
    {
        short key = (short)waitKey();
        if( key == 27 || key == 'q' || key == 'Q') // 'ESC'
            break;
    }

    return 0;
}


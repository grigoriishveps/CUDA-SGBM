#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "opencv2/calib3d.hpp"
#include "opencv2/imgproc.hpp"
#include "opencv2/imgcodecs.hpp"
#include "opencv2/highgui.hpp"
#include "opencv2/core/utility.hpp"
#include <iostream>
#include <bitset>
#include "./helpers/helper.cuh"
#include "./calc_cost/calc_cost.cuh"
#include "./calc_disparity/calc_disparity.cuh"
#include "./calc_path/calc_path.cuh"

using namespace cv;

using namespace cv::cuda;
using namespace std;

#define D_LVL 64

#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)

#define checkCudaErrors(call)                                 \
  do {                                                        \
    hipError_t err = call;                                   \
    if (err != hipSuccess) {                                 \
      printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, \
             hipGetErrorString(err));                        \
      exit(EXIT_FAILURE);                                     \
    }                                                         \
  } while (0)



__host__ void allProcessOnCUDA(unsigned char* census_l, unsigned char* census_r, int* pix_cost,  size_t rows, size_t cols) {
    int numBytes = rows * cols * D_LVL * sizeof(int);
    int smallBytes = rows * cols * D_LVL * sizeof(unsigned char);
    int extraBytes = rows * cols * D_LVL * sizeof(int);

    // allocate device memory
    unsigned char * adev = NULL, *bdev = NULL;
    int * extraStore = NULL;
    int * resCuda = NULL, *middleRes = NULL;

    checkCudaErrors(hipMalloc ( (void**)&adev, smallBytes ));
    checkCudaErrors(hipMalloc ( (void**)&bdev, smallBytes ));
    checkCudaErrors(hipMalloc ( (void**)&middleRes, numBytes ));
    checkCudaErrors(hipMalloc ( (void**)&extraStore, extraBytes ));
    checkCudaErrors(hipMalloc ( (void**)&resCuda, numBytes ));

    // set kernel launch configuration
  
    dim3 threads ( D_LVL );
    dim3 blocks  ( rows, cols );
    // create cuda event handles
    hipEvent_t start, stop;
    float gpuTime;
    float allRes = 0;
    int countCheck = 1;

    // TimeCheck
    for (int i = 0; i< countCheck ; i++) {
        gpuTime = 0.0f;
        checkCudaErrors(hipEventCreate ( &start ));
        checkCudaErrors(hipEventCreate ( &stop ));
        
        // asynchronously issue work to the GPU (all to stream 0)
        hipEventRecord ( start,  0 );
        checkCudaErrors(hipMemcpy( adev, census_l, smallBytes, hipMemcpyHostToDevice ));  // CAN ASYNC
        checkCudaErrors(hipMemcpy( bdev, census_r, smallBytes, hipMemcpyHostToDevice ));  // CAN ASYNC

        // COST
        processAgregateCostCUDA <<<blocks, threads>>> ( adev, bdev, middleRes, rows, cols);

        // PATH
        optimized_matMult_LEFT<<<rows, D_LVL>>> ( middleRes, extraStore, rows, cols);
        optimised_concatResCUDA<<<blocks, threads>>> (extraStore, resCuda, rows, cols);
        optimized_matMult_RIGHT<<<rows, D_LVL>>> (middleRes, extraStore, rows, cols);
        optimised_concatResCUDA<<<blocks, threads>>> (extraStore, resCuda, rows, cols);
        optimized_matMult_TOP<<<cols, D_LVL>>> (middleRes, extraStore, rows, cols);
        optimised_concatResCUDA<<<blocks, threads>>> (extraStore, resCuda, rows, cols);

        checkCudaErrors(hipMemcpy( pix_cost, resCuda, numBytes, hipMemcpyDeviceToHost ));
        hipEventRecord ( stop, 0 );

        hipEventSynchronize ( stop );
        hipEventElapsedTime ( &gpuTime, start, stop );

        hipEventDestroy ( start );
        hipEventDestroy ( stop  );
        allRes += gpuTime;
       
        // printf("Time spent executing by the GPU: %.3f millseconds\n", gpuTime);
    }
    
    printf("Average Time spent executing by the GPU: %.3f millseconds for COUNT=%d \n", allRes / countCheck, countCheck);

    // release resources
    checkCudaErrors(hipFree( adev  ));
    hipFree  ( bdev );
    hipFree  ( middleRes );
    hipFree  ( extraStore );
    hipFree  ( resCuda );
}


void calculateImageDisparity(cv::Mat &leftImage, cv::Mat &rightImage, cv::Mat *dispImg) {
    double costTime, disparityTime;
    size_t cols = leftImage.cols, rows = leftImage.rows;
    int *sum_cost = (int *) calloc(rows * cols * D_LVL, sizeof(int));

    if (!sum_cost) {
        printf("mem failure, exiting A \n");
        exit(EXIT_FAILURE);
    }

    unsigned char *census_l = (unsigned char *) calloc(rows * cols * D_LVL, sizeof(unsigned char));
    unsigned char *census_r = (unsigned char *) calloc(rows * cols * D_LVL, sizeof(unsigned char));

    // cout << "1. Census Transform" << endl;
    census_transform(leftImage, census_l, rows, cols);
    census_transform(rightImage, census_r, rows, cols);

    // 2. Calculate Pixel Cost.
    // 3. Aggregate Cost
    //One CUDA operation
    costTime = (double) getTickCount();
    allProcessOnCUDA(census_l, census_r, sum_cost, rows, cols);
    costTime = ((double)getTickCount() - costTime)/getTickFrequency();

    // 4. Create Disparity Image.
    cout << "3. Create Disparity Image." << endl;
    disparityTime = (double) getTickCount();
    calc_disparity(sum_cost, *dispImg, rows, cols);
    disparityTime = ((double)getTickCount() - disparityTime)/getTickFrequency();

    cout<<"Cost algorithm time: "<< costTime <<"s"<<endl;  // 120ms
    cout<<"Disparity algorithm time: "<< disparityTime <<"s"<<endl;  // 36ms

    free(sum_cost);
    free(census_l);
    free(census_r);
}


int main () {
    double solving_time, allTimeSolving = (double) getTickCount();

    Mat leftImage = cv::imread("./src/images/leftImage1.png",cv::IMREAD_GRAYSCALE);
    Mat rightImage = cv::imread("./src/images/rightImage1.png",cv::IMREAD_GRAYSCALE);
    // Mat leftImage = cv::imread("./src/images/appleLeft.png",cv::IMREAD_GRAYSCALE);
    // Mat rightImage = cv::imread("./src/images/appleRight.png",cv::IMREAD_GRAYSCALE);

    size_t cols = leftImage.cols, rows = leftImage.rows;
    cv::Mat disparityMap, *dispImg = new cv::Mat(rows, cols, CV_8UC1);

    cout.precision(3);
    cout << " Start timing"<< endl;
    solving_time = (double) getTickCount();
    // resize(leftImage, left_for_matcher, Size(),0.1,0.1, INTER_LINEAR_EXACT);
    // cvtColor(left_for_matcher,  left_for_matcher,  COLOR_BGR2GRAY);
    // left_for_matcher.convertTo(left_for_matcher, CV_16UC1);
    // leftImage.convertTo(leftImage, CV_16UC3);
    
    // Mat splitResult[3];
    // split(leftImage, splitResult);
    // Mat leftImageR = splitResult[0];
    // Mat leftImageG = splitResult[1];
    // Mat leftImageB = splitResult[2];

    // imshow("Blue Channel", leftImageR);//showing Blue channel//
    // imshow("Green Channel", leftImageG);//showing Green channel//
    // imshow("Red Channel", leftImageB);
    // cout << leftImageR;

    // short** arrayLeftImageR = toMatArray(splitResult[0]);
    // short** arrayLeftImageG = toMatArray(splitResult[1]);
    // short** arrayLeftImageB = toMatArray(splitResult[2]);

    // short** arrayRightImageR = toMatArray(splitResult[0]);
    // short** arrayRightImageG = toMatArray(splitResult[1]);
    // short** arrayRightImageB = toMatArray(splitResult[2]);

    calculateImageDisparity(leftImage, rightImage, dispImg);
  


    // Visualize Disparity Image.
    disparityMap = *dispImg;
    disparityMap.convertTo(disparityMap, CV_8U, 256.0/D_LVL);
    applyColorMap(disparityMap, disparityMap, COLORMAP_JET);
    imshow("leftImage", disparityMap);

    // END TEST GRAYSCALE

    solving_time = ((double)getTickCount() - solving_time)/getTickFrequency();
    allTimeSolving = ((double)getTickCount() - allTimeSolving)/getTickFrequency();

    cout<<"Process time: "<<solving_time<<"s"<<endl;     // 179ms
    cout<<"All run time: "<<allTimeSolving<<"s"<<endl;   // 184ms
    std::cout << "OK"<< std::endl;

    free(dispImg);

    while(1)
    {
        short key = (short)waitKey();
        if( key == 27 || key == 'q' || key == 'Q') // 'ESC'
            break;
    }

    return 0;
}


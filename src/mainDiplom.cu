#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "opencv2/calib3d.hpp"
#include "opencv2/imgproc.hpp"
#include "opencv2/imgcodecs.hpp"
#include "opencv2/highgui.hpp"
#include "opencv2/core/utility.hpp"
#include <iostream>
#include <bitset>
#include "./helpers/helper.cuh"
#include "./calc_cost/calc_cost_and_disparity.cuh"
#include "./calc_path/calc_path.cuh"

using namespace cv;

using namespace cv::cuda;
using namespace std;

#define D_LVL 64

#define checkCudaErrors(call)                                 \
  do {                                                        \
    hipError_t err = call;                                   \
    if (err != hipSuccess) {                                 \
      printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, \
             hipGetErrorString(err));                        \
      exit(EXIT_FAILURE);                                     \
    }                                                         \
  } while (0)



__host__ void allProcessOnCUDA(
    unsigned char* census_l_R, unsigned char* census_l_G, unsigned char* census_l_B,
    unsigned char* census_r_R, unsigned char* census_r_G, unsigned char* census_r_B,
    uchar* disparityMap,
    size_t rows, size_t cols
) {
    int numBytes = rows * cols * D_LVL * sizeof(int);
    int smallBytes = rows * cols * D_LVL * sizeof(unsigned char);

    // allocate device memory
    unsigned char * adev = NULL, *bdev = NULL, *disparityRes = NULL;
    unsigned char * adevRes = NULL, *bdevRes = NULL;
    int * extraStore = NULL;
    int * resCuda = NULL, *middleRes = NULL;

    checkCudaErrors(hipMalloc ( (void**)&adev, smallBytes ));
    checkCudaErrors(hipMalloc ( (void**)&bdev, smallBytes ));
    checkCudaErrors(hipMalloc ( (void**)&adevRes, smallBytes ));
    checkCudaErrors(hipMalloc ( (void**)&bdevRes, smallBytes ));
    checkCudaErrors(hipMalloc ( (void**)&middleRes, numBytes ));
    checkCudaErrors(hipMalloc ( (void**)&extraStore, numBytes ));
    checkCudaErrors(hipMalloc ( (void**)&resCuda, numBytes  ));
    checkCudaErrors(hipMalloc ( (void**)&disparityRes, smallBytes));


    // hipHostRegister(adev, smallBytes, 0);
    // hipHostRegister(bdev, smallBytes, 1);


    // set kernel launch configuration
  
    dim3 threads ( D_LVL );
    dim3 blocks  ( rows, cols );
    dim3 miniBlocks  ( rows - 2, cols - 2 );
    hipStream_t stream1, stream2, stream3;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);

    // create cuda event handles
    hipEvent_t start, stop;
    float gpuTime;
    float allRes = 0;
    int countCheck = 1;

    // TimeCheck
    for (int i = 0; i< countCheck ; i++) {
        gpuTime = 0.0f;
        checkCudaErrors(hipEventCreate ( &start ));
        checkCudaErrors(hipEventCreate ( &stop ));
        
        // asynchronously issue work to the GPU (all to stream 0)
        hipEventRecord ( start,  0 );

        clearResCUDA<<<blocks, threads>>> (middleRes, rows, cols);
        clearResCUDA<<<blocks, threads>>> (resCuda, rows, cols);
        clearResCUDA<<<blocks, threads>>> (disparityRes, rows, cols);


        // COST
        checkCudaErrors(hipMemcpy( adev, census_l_R, smallBytes, hipMemcpyHostToDevice ));
        checkCudaErrors(hipMemcpy( bdev, census_r_R, smallBytes, hipMemcpyHostToDevice ));
        censusTransform<<<miniBlocks, 1>>> (adev, adevRes, cols);
        censusTransform<<<miniBlocks, 1>>> (bdev, bdevRes, cols);
        calculateInitialCostCUDA <<<blocks, threads>>> ( adevRes, bdevRes, extraStore, rows, cols);
        optimisedConcatResCUDA<<<blocks, threads>>> (extraStore, middleRes, rows, cols);

        hipDeviceSynchronize();
        checkCudaErrors(hipMemcpy( adev, census_l_G, smallBytes, hipMemcpyHostToDevice ));
        checkCudaErrors(hipMemcpy( bdev, census_r_G, smallBytes, hipMemcpyHostToDevice ));
        censusTransform<<<miniBlocks, 1>>> (adev, adevRes, cols);
        censusTransform<<<miniBlocks, 1>>> (bdev, bdevRes, cols);
        calculateInitialCostCUDA <<<blocks, threads>>> ( adevRes, bdevRes, extraStore, rows, cols);
        optimisedConcatResCUDA<<<blocks, threads>>> (extraStore, middleRes, rows, cols);
    
        checkCudaErrors(hipMemcpy( adev, census_l_B, smallBytes, hipMemcpyHostToDevice ));
        checkCudaErrors(hipMemcpy( bdev, census_r_B, smallBytes, hipMemcpyHostToDevice ));
        censusTransform<<<miniBlocks, 1>>> (adev, adevRes, cols);
        censusTransform<<<miniBlocks, 1>>> (bdev, bdevRes, cols);
        calculateInitialCostCUDA <<<blocks, threads>>> ( adevRes, bdevRes, extraStore, rows, cols);
        optimisedConcatResCUDA<<<blocks, threads>>> (extraStore, middleRes, rows, cols);


        // PATH
        calculatePathLeft<<<rows, D_LVL>>> ( middleRes, extraStore, rows, cols);
        optimisedConcatResCUDA<<<blocks, threads>>> (extraStore, resCuda, rows, cols);
        calculatePathRight<<<rows, D_LVL>>> (middleRes, extraStore, rows, cols);
        optimisedConcatResCUDA<<<blocks, threads>>> (extraStore, resCuda, rows, cols);
        calculatePathTop<<<cols, D_LVL>>> (middleRes, extraStore, rows, cols);
        optimisedConcatResCUDA<<<blocks, threads>>> (extraStore, resCuda, rows, cols);
        calculatePathBackslash<<<cols + rows - 1, D_LVL>>> (middleRes, extraStore, rows, cols);
        optimisedConcatResCUDA<<<blocks, threads>>> (extraStore, resCuda, rows, cols);
        calculatePathSlash<<<cols + rows - 1, D_LVL>>> (middleRes, extraStore, rows, cols);
        optimisedConcatResCUDA<<<blocks, threads>>> (extraStore, resCuda, rows, cols);


        //Disparyty
        calculateDisparityCUDA<<<blocks, 1>>> (resCuda, disparityRes, cols);

        checkCudaErrors(hipMemcpy( disparityMap, disparityRes, smallBytes, hipMemcpyDeviceToHost ));
        
        hipEventRecord ( stop, 0 );

        hipEventSynchronize ( stop );
        hipEventElapsedTime ( &gpuTime, start, stop );

        hipStreamDestroy(stream1);
        hipStreamDestroy(stream2);
        hipStreamDestroy(stream3);

        hipEventDestroy ( start );
        hipEventDestroy ( stop  );
        allRes += gpuTime;
       
        // printf("Time spent executing by the GPU: %.3f millseconds\n", gpuTime);
    }
    
    printf("Average Time spent executing by the GPU: %.3f millseconds for COUNT=%d \n", allRes / countCheck, countCheck);

    // release resources
    checkCudaErrors(hipFree( adev  ));
    hipFree  ( bdev );
    checkCudaErrors(hipFree( adevRes  ));
    hipFree  ( bdevRes );
    hipFree  ( middleRes );
    hipFree  ( extraStore );
    hipFree  ( resCuda );
    hipFree  ( disparityRes );
}


__host__ cv::Mat* calculateImageDisparity(cv::Mat &leftImage, cv::Mat &rightImage) {
    double costTime;
    size_t cols = leftImage.cols, rows = leftImage.rows;
    uchar *disparityMap = (uchar *) calloc(rows * cols * D_LVL, sizeof(uchar));

    if (!disparityMap) {
        printf("mem failure, exiting A \n");
        exit(EXIT_FAILURE);
    }


    Mat splitResultLeft[3], splitResultRight[3];
    split(leftImage, splitResultLeft);
    split(rightImage, splitResultRight);

    costTime = (double) getTickCount();
    int sizeMem = rows * cols * D_LVL * sizeof(unsigned char);

    // unsigned char *census_l_R;
    // unsigned char *census_l_G;
    // unsigned char *census_l_B;
    // unsigned char *census_r_R;
    // unsigned char *census_r_G;
    // unsigned char *census_r_B;

    unsigned char *census_l_R = splitResultLeft[0].data;
    unsigned char *census_l_G = splitResultLeft[1].data;
    unsigned char *census_l_B = splitResultLeft[2].data;
    unsigned char *census_r_R = splitResultRight[0].data;
    unsigned char *census_r_G = splitResultRight[1].data;
    unsigned char *census_r_B = splitResultRight[2].data;

    hipHostRegister(&census_l_R, sizeMem, 0);
    hipHostRegister(&census_l_G, sizeMem, 0);
    hipHostRegister(&census_l_B, sizeMem, 0);
    hipHostRegister(&census_r_R, sizeMem, 0);
    hipHostRegister(&census_r_G, sizeMem, 0);
    hipHostRegister(&census_r_B, sizeMem, 0);

    // 1. Census Transform"
    // 2. Calculate Pixel Cost.
    // 3. Aggregate Cost
    // 4. Create Disparity Image.
    //One CUDA operation
    
    allProcessOnCUDA(census_l_R, census_l_G, census_l_B, census_r_R, census_r_G, census_r_B, disparityMap, rows, cols);
    costTime = ((double)getTickCount() - costTime)/getTickFrequency();

    cout<<"Cost algorithm time: "<< costTime <<"s"<<endl;  // 123ms

    hipHostUnregister(census_l_R);
    hipHostUnregister(census_r_G);
    hipHostUnregister(census_l_B);
    hipHostUnregister(census_r_R);
    hipHostUnregister(census_l_G);
    hipHostUnregister(census_r_B);
    

    return new Mat(rows, cols, CV_8UC1, disparityMap);
}


int main () {
    double solving_time, allTimeSolving = (double) getTickCount();
    // Mat leftImage = cv::imread("./src/images/leftImage1.png",cv::IMREAD_GRAYSCALE);
    // Mat rightImage = cv::imread("./src/images/rightImage1.png",cv::IMREAD_GRAYSCALE);
    // Mat leftImage = cv::imread("./src/images/leftImage1.png",cv::IMREAD_COLOR);
    // Mat rightImage = cv::imread("./src/images/rightImage1.png",cv::IMREAD_COLOR);
    // Mat leftImage = cv::imread("./src/images/appleLeft.jpg");
    // Mat rightImage = cv::imread("./src/images/appleRight.jpg");
    // Mat leftImage = cv::imread("./src/images/warLeft.jpg",cv::IMREAD_COLOR);
    // Mat rightImage = cv::imread("./src/images/warRight.jpg",cv::IMREAD_COLOR);
    Mat leftImage = cv::imread("./src/images/warLeft.jpg");
    Mat rightImage = cv::imread("./src/images/warRight.jpg");
    // Mat leftImage = cv::imread("./src/images/warLeft.jpg",cv::IMREAD_GRAYSCALE);
    // Mat rightImage = cv::imread("./src/images/warRight.jpg",cv::IMREAD_GRAYSCALE);

    // imshow("leftImage", leftImage);
    // imshow("rightImage", rightImage);

    cv::Mat *disparityMap;

    cout.precision(3);
    cout << " Start timing"<< endl;
    solving_time = (double) getTickCount();

    // resize(leftImage, left_for_matcher, Size(),0.1,0.1, INTER_LINEAR_EXACT);
    // cvtColor(left_for_matcher,  left_for_matcher,  COLOR_BGR2GRAY);
    // left_for_matcher.convertTo(left_for_matcher, CV_16UC1);
    // leftImage.convertTo(leftImage, CV_16UC3);
    disparityMap = calculateImageDisparity(leftImage, rightImage);




    // END TEST GRAYSCALE
    solving_time = ((double)getTickCount() - solving_time)/getTickFrequency();
 

    // Visualize Disparity Image.
    disparityMap->convertTo(*disparityMap, CV_8U, 256.0/D_LVL);
    applyColorMap(*disparityMap, *disparityMap, COLORMAP_JET);
    imshow("disparityMap", *disparityMap);

    allTimeSolving = ((double)getTickCount() - allTimeSolving)/getTickFrequency();
    cout<<"Process time: "<<solving_time<<"s"<<endl;     // 179ms
    cout<<"All run time: "<<allTimeSolving<<"s"<<endl;   // 184ms
    std::cout << "OK"<< std::endl;

    free(disparityMap);

    while(1)
    {
        short key = (short)waitKey();
        if( key == 27 || key == 'q' || key == 'Q') // 'ESC'
            break;
    }

    return 0;
}


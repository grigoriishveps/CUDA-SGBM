#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "opencv2/calib3d.hpp"
#include "opencv2/imgproc.hpp"
#include "opencv2/imgcodecs.hpp"
#include "opencv2/highgui.hpp"
#include "opencv2/core/utility.hpp"
#include <iostream>
#include <bitset>
#include "./helpers/helper.cuh"
#include "./calc_cost/calc_cost.cuh"
#include "./calc_disparity/calc_disparity.cuh"
#include "./calc_path/calc_path.cuh"

using namespace cv;

using namespace cv::cuda;
using namespace std;

#define D_LVL 64


int main () {
    double solving_time, allTimeSolving = (double) getTickCount();

    Mat leftImage = cv::imread("./src/images/leftImage1.png",cv::IMREAD_GRAYSCALE);
    Mat rightImage = cv::imread("./src/images/rightImage1.png",cv::IMREAD_GRAYSCALE);

    size_t cols = leftImage.cols, rows = leftImage.rows;
    cv::Mat disparityMap, *dispImg = new cv::Mat(rows, cols, CV_8UC1);

    cout << " Start timing"<< endl;
    solving_time = (double) getTickCount();
    // resize(leftImage, left_for_matcher, Size(),0.1,0.1, INTER_LINEAR_EXACT);
    // cvtColor(left_for_matcher,  left_for_matcher,  COLOR_BGR2GRAY);
    // left_for_matcher.convertTo(left_for_matcher, CV_16UC1);
    // leftImage.convertTo(leftImage, CV_16UC3);
    
    // short leftImageColors =

    // Mat splitResult[3];
    // split(leftImage, splitResult);
    // Mat leftImageR = splitResult[0];
    // Mat leftImageG = splitResult[1];
    // Mat leftImageB = splitResult[2];

    // imshow("Blue Channel", leftImageR);//showing Blue channel//
    // imshow("Green Channel", leftImageG);//showing Green channel//
    // imshow("Red Channel", leftImageB);
    // cout << leftImageR;

    // short** arrayLeftImageR = toMatArray(splitResult[0]);
    // short** arrayLeftImageG = toMatArray(splitResult[1]);
    // short** arrayLeftImageB = toMatArray(splitResult[2]);

    // short** arrayRightImageR = toMatArray(splitResult[0]);
    // short** arrayRightImageG = toMatArray(splitResult[1]);
    // short** arrayRightImageB = toMatArray(splitResult[2]);

    // checkCSCT(leftImage);

    // TEST GRAYSCALE

    // // 1. Census Transform.
    // cout << "1. Census Transform" << endl;
    
    // short** arrayLeftImage = toMatArray(leftImage);
    // short** arrayRightImage = toMatArray(rightImage);

    // uint **CSCTLeftImage = toCSCT(arrayLeftImage, rows, cols);
    // uint **CSCTRightImage = toCSCT(arrayRightImage, rows, cols);


    double costTime, pathTime, disparityTime;
    int *pix_cost = (int *) calloc(rows * cols * D_LVL, sizeof(int));
    int *sum_cost = (int *) calloc(rows * cols * D_LVL, sizeof(int));
    
    if (!pix_cost || !sum_cost) {
        printf("mem failure, exiting A \n");
        exit(EXIT_FAILURE);
    }

    cv::Mat *census_l = new cv::Mat(rows, cols, CV_8UC1);
    cv::Mat *census_r = new cv::Mat(rows, cols, CV_8UC1);

    // cout << "1. Census Transform" << endl;
    census_transform(leftImage, *census_l, rows, cols);
    census_transform(rightImage, *census_r, rows, cols);

    // 2. Calculate Pixel Cost.
    cout << "1. Calculate Pixel Cost." << endl;
    costTime = (double) getTickCount();

    calcCost_CUDA(*census_l, *census_r, pix_cost, rows, cols);
    costTime = ((double)getTickCount() - costTime)/getTickFrequency();


    // 3. Aggregate Cost
    cout << "2. Aggregate Cost" << endl;
    pathTime = (double) getTickCount();
    optimized_agregateCostCUDA(pix_cost, sum_cost, rows, cols);   // 20ms
    pathTime = ((double)getTickCount() - pathTime)/getTickFrequency();


    // 4. Create Disparity Image.
    cout << "3. Create Disparity Image." << endl;
    disparityTime = (double) getTickCount();
    calc_disparity(sum_cost, *dispImg, rows, cols);
    disparityTime = ((double)getTickCount() - disparityTime)/getTickFrequency();




    // Visualize Disparity Image.
    disparityMap = *dispImg;
    disparityMap.convertTo(disparityMap, CV_8U, 256.0/D_LVL);
    applyColorMap(disparityMap, disparityMap, COLORMAP_JET);
    imshow("leftImage", disparityMap);

    // END TEST GRAYSCALE

    solving_time = ((double)getTickCount() - solving_time)/getTickFrequency();
    allTimeSolving = ((double)getTickCount() - allTimeSolving)/getTickFrequency();
    cout.precision(3);
    cout<<"Cost algorithm time: "<< costTime <<"s"<<endl;  // 120ms
    cout<<"Path algorithm time: "<< pathTime <<"s"<<endl;  // 48ms
    cout<<"Disparity algorithm time: "<< disparityTime <<"s"<<endl;  // 36ms
    cout<<"Process time: "<<solving_time<<"s"<<endl;     // 230ms
    cout<<"All run time: "<<allTimeSolving<<"s"<<endl;   // 233ms
    std::cout << "OK"<< std::endl;
    while(1)
    {
        short key = (short)waitKey();
        if( key == 27 || key == 'q' || key == 'Q') // 'ESC'
            break;
    }

    return 0;
}

